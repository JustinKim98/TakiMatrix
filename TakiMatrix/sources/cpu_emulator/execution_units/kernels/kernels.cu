#include "hip/hip_runtime.h"
//
// Created by jwkim98 on 19/02/02.
//

#include "../../../../includes/cpu_emulator/execution_units/kernels/kernels.h"

namespace kernel {
    __global__ void add_kernel(float* first, float* second, size_t size)
    {
        size_t compute_stride = blockDim.x*gridDim.x;
        size_t compute_index = blockIdx.x*blockDim.x+threadIdx.x;

        while (compute_index<size) {
            second[compute_index] = first[compute_index]+second[compute_index];
            /// for cases when matrix size exceeds compute_index
            compute_index += compute_stride;
        }
    }

    __global__ void sub_kernel(float* first, float* second, size_t size)
    {
        size_t compute_stride = blockDim.x*gridDim.x;
        size_t compute_index = blockIdx.x*blockDim.x+threadIdx.x;

        while (compute_index<size) {
            second[compute_index] = first[compute_index]-second[compute_index];
            /// for cases when matrix size exceeds compute_index
            compute_index += compute_stride;
        }
    }

    __global__ void mul_kernel(float* first, float* second, float* result,
            size_t middle_size, size_t first_row_num,
            size_t second_col_num, size_t dimension_num)
    {
        /// result will have first_row rows and second_col columns
        size_t col_dim = first_row_num;
        size_t row_dim = second_col_num;
        size_t dim_size = col_dim*row_dim;
        size_t size = col_dim*row_dim*dimension_num;

        size_t compute_stride = blockDim.x*gridDim.x;
        size_t compute_index = blockIdx.x*blockDim.x+threadIdx.x;

        while (compute_index<size) {
            float sum = 0;
            size_t compute_row = compute_index/row_dim;
            size_t compute_col = compute_index%row_dim;
            size_t compute_dim = compute_index/dim_size;

            for (int count = 0; count<middle_size; count++) {
                sum += first[compute_row*col_dim+count]*
                        second[compute_col+count*row_dim + compute_dim*dim_size];
            }
            result[compute_row*row_dim+compute_col] = sum;
            compute_index += compute_stride;
        }
    }

    template<typename Func>
    __global__ void dot_kernel(float* data, Func func, size_t size)
    {
        size_t compute_stride = blockDim.x*gridDim.x;
        size_t compute_index = blockIdx.x*blockDim.x+threadIdx.x;

        while (compute_index<size) {
            data[compute_index] = func(data);
            /// for cases when matrix size exceeds compute_index
            compute_index += compute_stride;
        }
    }
} // namespace kernel
