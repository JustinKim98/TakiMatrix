#include "hip/hip_runtime.h"
#include "../includes/test.h"
#include "hip/hip_runtime.h"
#include ""
#include "thrust/device_vector.h"
#include "thrust/transform.h"
#include <iostream>

__global__ void add_kernel(int *a, int *b, int size)
{
    if(threadIdx.x < size){
        b[threadIdx.x] =  a[threadIdx.x] + b[threadIdx.x];
    }
}


void add_with_cuda(){
    int a[5] = {1,2,3,4,5};
    int b[5] = {1,2,3,4,5};


    int *device_a;
    int *device_b;
    int *device_c;

    hipMalloc((void**)&device_a, 5*sizeof(int));
    hipMalloc((void**)&device_b, 5*sizeof(int));
    hipMemcpy(device_a, a, 5*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_b, b, 5*sizeof(int), hipMemcpyHostToDevice);
    add_kernel<<<1,100>>>(device_a, device_b, 5);
    hipMemcpy(a, device_a, 5*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(b, device_b, 5*sizeof(int), hipMemcpyDeviceToHost);

    for(int count = 0; count < 5; count++){
        std::cout<<b[count]<<" ";
    }
    hipFree(device_a);
    hipFree(device_b);
}